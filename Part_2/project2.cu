#include "hip/hip_runtime.h"
#pragma optimize( "", off )


#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <memory.h>
#include <malloc.h>
#include <string>
#include <iostream>
#include <fstream>
#include <chrono>

using namespace std;

// Example data to load from your file:
// 117,85,146,194,21,20,20,20,20,20,20,20,20,20,20,20,20,
// 20,20,20,20,20,20,20,21,22,417,418,141,68,196,198,194,177,
// 173,173,172,2101,172,172,173,149,172,172,172,173,172,175,
// 173,173,172,171,172,100,111,101,101,100,98,98,98,88,
// 98,99,97,98,96,96,97,98,98,96,98,98,97,98,97,97,92,96


// A few filtering kernels as samples
float low_pass_kernel[9] = {
    1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0,
    1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0,
    1.0 / 9.0, 1.0 / 9.0, 1.0 / 9.0
};

// float the inputs
float hor_line_kernel[9] = {
    -1.0, -1.0, -1.0,
     2.0,  2.0,  2.0,
    -1.0, -1.0, -1.0
};

float ver_line_kernel[9] = {
    -1.0, 2.0, -1.0,
    -1.0, 2.0, -1.0,
    -1.0, 2.0, -1.0
};

__constant__ float convKernal[9];

__global__ void convolution(int* distArray, float* result, int rowIndex, int colIndex, int maskIndex, int calcAmount) {
    // Global thread positions
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int amt = blockIdx.z * blockDim.z + threadIdx.z;

    // Calculate radius of the mask
    int r = maskIndex / 2;

    // Calculate the start point for the element
    int startcol = col - r;
    int startrow = row - r;

    // Temp value for calculation
    float temp = 0;

    // Execute function 1000 times
    if (amt >= 0 && amt < calcAmount) {
        //printf("Thread: %d", amt);
        if (row < rowIndex && col < colIndex) {
            // go over each element of the mask
            for (int i = 0; i < maskIndex; i++) {
                for (int j = 0; j < maskIndex; j++) {

                    // Calculate convolution if convolution matrix fits into the current row/col
                    if (startcol >= 0 && col < (colIndex - r)) {
                        if (startrow >= 0 && row < (rowIndex - r)) {
                            temp += convKernal[i * maskIndex + j] * distArray[(row - (i - 1)) * colIndex + (col - (j - 1))];
                            result[row * colIndex + col] = temp / 255;
                        }

                    }

                }
            }
        }
    }
}


void read_file(string filename, int* output_array, int data_count, string data_delimiter) {
    fstream data_file;
    char file_character;
    int temp_int = 0;
    string temp_str;
    data_file.open(filename.c_str(), ios::in);

    if (data_file.is_open()) {
        while (data_file.good()) {
            for (int i = 0; i < data_count; i++) {
                temp_int = 0;
                temp_str = "";
                data_file.get(file_character);
                while (file_character != data_delimiter[0]) {
                    temp_str += file_character;
                    data_file.get(file_character);
                }
                temp_int = stoi(temp_str);
                output_array[i] = temp_int;
            }
            data_file.close();
            break;
        }
    }
}


int main(int argc, char* argv[]) {

    clock_t start, end;
    double cpu_time_used;

    if (argc < 2) {
        printf("Need 2 arguments! X(Number of positions) and Y(Max Distance)\n\n");
        return -1;
    }


    int posNum = atoi(argv[1]);
    int dstNum = atoi(argv[2]);
    printf("Positions: %d, Max Distance: %d\n", posNum, dstNum);

    int* distance_vector = (int*)calloc(posNum, sizeof(int));
    int* distance_matrix = (int*)calloc(posNum * dstNum, sizeof(int));
    float* filtered_matrix = (float*)calloc(posNum * dstNum, sizeof(float));
    float* filtered_matrix_cpu = (float*)calloc(posNum * dstNum, sizeof(float));
    int* threshold_matrix = (int*)calloc(posNum * dstNum, sizeof(int));
    int* new_vector = (int*)calloc(posNum, sizeof(int));

    int i;


    // Implement your LOAD_DATA function here to load X number of elements and store them into distance_vector
    read_file("data.txt", distance_vector, posNum, ",");

    for (int d = 0; d < posNum; d++) {
        printf("distance_vector[%d]: %d\n", d, distance_vector[d]);
    }


    // Creates matrix from input vector
    for (i = 0; i < posNum; i++) {
        int distance = distance_vector[i];
        if (distance >= dstNum) distance = dstNum - 1;
        distance_matrix[distance * posNum + i] = 255; //sets distance object
    }

    // Start time measure
    start = clock();
    auto start_chrono = chrono::steady_clock::now();

    ///******************* OPTIMIZE THIS ***********************/

    // Number of iterations of the calculations 
    int calcAmount = 1000;

    // Number of elements in indexed matrixed array
    int n = posNum * dstNum;

    // Bytes of indexed matrixed array
    int bytes_n = n * sizeof(int);

    // Bytes of output matrix array
    int bytes_out = n * sizeof(float);

    // Size of convolution mask (indexed matrix array)
    int maskIndex = 3;

    // Size of convolution mask in bytes
    int bytes_maskIndex = (maskIndex * maskIndex) * sizeof(float);

    // Allocate space on the device 
    float* d_hor_line_kernel, * d_filtered_matrix;
    int* d_distance_matrix;

    //hipMalloc(&d_hor_line_kernel, bytes_maskIndex);
    hipMalloc(&d_filtered_matrix, bytes_out);
    hipMalloc(&d_distance_matrix, bytes_n);

    // Copy the data to the device
    //hipMemcpy(d_hor_line_kernel, hor_line_kernel, bytes_maskIndex, hipMemcpyHostToDevice);
    hipMemcpy(d_distance_matrix, distance_matrix, bytes_n, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(convKernal), hor_line_kernel, bytes_maskIndex);

    // Threads per Threadblock (TB)
    int THREADS = 8;

    // Dimension arguments
    dim3 block_dim(THREADS, THREADS, THREADS);
    dim3 grid_dim(16, 64, 128);

    auto end_chrono = chrono::steady_clock::now();
    cout << "GPU Data Transfer time: " << chrono::duration_cast<chrono::milliseconds>(end_chrono - start_chrono).count() << " ms" << endl;

    // Call the kernel
    start_chrono = chrono::steady_clock::now();
    convolution << <grid_dim, block_dim >> > (d_distance_matrix, d_filtered_matrix, dstNum, posNum, maskIndex, calcAmount);

    // Wait for threads to finish calculations
    hipDeviceSynchronize();
    end_chrono = chrono::steady_clock::now();
    cout << "GPU Calculation time: " << chrono::duration_cast<chrono::microseconds>(end_chrono - start_chrono).count() << " us" << endl;


    // Copy back the result
    hipMemcpy(filtered_matrix, d_filtered_matrix, bytes_out, hipMemcpyDeviceToHost);



    int l, j, k, x, y;
    float sum = 0.0;

    // Wait for threads to finish calculations

   // Repeat 1000 times
    start_chrono = chrono::steady_clock::now();
    for (l = 0; l < 1000; l++) {

        // Apply kernel for all points in the matrix
        for (y = 1; y < dstNum - 1; y++) {
            for (x = 1; x < posNum - 1; x++) {
                sum = 0.0;
                for (k = -1; k < 2; k++) {
                    for (j = -1; j < 2; j++) {
                        sum += hor_line_kernel[(k + 1) * 3 + (j + 1)] * (float)distance_matrix[(y - k) * posNum + (x - j)];
                    }
                }
                filtered_matrix_cpu[y * posNum + x] = sum / 255;
            }
        }
    }
    end_chrono = chrono::steady_clock::now();
    cout << "CPU Calculation time: " << chrono::duration_cast<chrono::milliseconds>(end_chrono - start_chrono).count() << " ms" << endl;
    ///********************************************************/
    // Print arrays
    for (int i = 0; i < dstNum * posNum; i++) {
        if (filtered_matrix_cpu[i] != filtered_matrix[i]) {
            printf("ERROR: [%d] CPU: %f | GPU: %f\n", i, filtered_matrix_cpu[i], filtered_matrix[i]);
        }
        else
        {
            // printf("[%d] CPU & GPU: %f\n", i, filtered_matrix[i]);
        }

    }

    // End time measure
    end = clock();
    cpu_time_used = ((double)(end - start) / 1000) / CLOCKS_PER_SEC;

    // Threshold the matrix
    for (x = 0; x < posNum; x++) {
        for (y = 0; y < dstNum; y++) {
            if (filtered_matrix[y * posNum + x] >= 4.0) {
                threshold_matrix[y * posNum + x] = 1;
            }
        }
    }

    // Extract vector from matrix
    for (x = 0; x < posNum; x++) {
        for (y = 0; y < dstNum; y++) {
            if (threshold_matrix[y * posNum + x]) {
                new_vector[x] = y;//sets distance object
            }
        }
        if (new_vector[x] == 0) new_vector[x] = 300;
    }

    // Prints threshold-filtered vector
    for (x = 0; x < posNum; x++) {
        printf("%d, ", new_vector[x]);
    }

    // printf("\nTotal time = %f ms\n", cpu_time_used * 1000);


    free(distance_vector);
    free(distance_matrix);
    free(filtered_matrix);
    free(threshold_matrix);
    free(new_vector);

    return 0;
}
